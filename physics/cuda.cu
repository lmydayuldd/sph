#include "hip/hip_runtime.h"
#ifndef CUDA_CU_H
#define CUDA_CU_H

#include <hip/hip_runtime.h>
#include <builtin_types.h> // __global__ __host__ __device__

#define UNIFIED_MATH_CUDA_H
#include <vector_functions.h>
#include <math_functions.h>
#include <hip/hip_runtime.h> // hipMalloc(), hipMemcpy(), ...

//#include <thrust/device_vector.h>
//#include <thrust/host_vector.h>

#include <vector>
//#ifdef COMPILER_MSVC
#include <algorithm>
//#endif

#include "machine/particle.h"
#include "physics/forces.h"
#include "physics/vector.h"
#include "util/settings.h"

using namespace std;

////////////////////////////////////////////////////////////////////////////////
//// Interface for outside world. //////////////////////////////////////////////
//////// Kernel (= global = host + device) functions. //////////////////////////
////////////////////////////////////////////////////////////////////////////////

__forceinline __device__ double3 operator-(const double3 &v1, const double3 &v2) {
    return make_double3(v1.x-v2.x, v1.y-v2.y, v1.z-v2.z);
}

__forceinline __device__ double3 operator+(const double3 &v1, const double3 &v2) {
    return make_double3(v1.x+v2.x, v1.y+v2.y, v1.z+v2.z);
}

__forceinline __device__ double3 operator/(const double3 &v, const double &x) {
    return make_double3(v.x/x, v.y/x, v.z/x);
}

__forceinline __device__ double3 operator*(const double3 &v, const double &x) {
    return make_double3(v.x*x, v.y*x, v.z*x);
}

__forceinline __device__ double dot(const double3 &v1, const double3 &v2) {
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__forceinline __device__ double norm(const double3 &v) {
    return norm3d(v.x, v.y, v.z);
}

__forceinline __device__ double distance(const double3 &v1, const double3 &v2) {
    double3 v3 = v1 - v2;
    return norm(v3);
}

__forceinline __device__ double3 normal(const double3 &v) {
    return v / norm(v);
}

__forceinline __device__ void collidePair(double &rx1, double &ry1, double &rz1,
                                          double &vx1, double &vy1, double &vz1,
                                          double &rx2, double &ry2, double &rz2,
                                          double &vx2, double &vy2, double &vz2,
                                          double &m1, double &m2,
                                          double &radius,
                                          unsigned &stationary1, unsigned &stationary2)
{
    //double3 sphereNormal;
    double3 p1_r = make_double3(rx1, ry1, rz1);
    double3 p2_r = make_double3(rx2, ry2, rz2);
    double3 p1_v = make_double3(vx1, vy1, vz1);
    double3 p2_v = make_double3(vx2, vy2, vz2);
    double3 sphereNormal = normal(p1_r - p2_r);
    double distanceBorder = distance(p1_r, p2_r) - radius - radius;
    if (distanceBorder < 0.) {
//        if (! stationary2)
        {
            p2_r = p2_r + sphereNormal * distanceBorder;
        }

        double3 p2_v_old = make_double3(p2_v.x, p2_v.y, p2_v.z);
//        if (! stationary2)
        {
            p2_v = p2_v -
                (p2_r - p1_r) * (
                    dot(p2_v - p1_v, p2_r - p1_r)
                    / pow(distance(p2_r, p1_r), 2)
                    * 2 * m1 / (m1 + m2)
                );
    //                    *p2.v *= (1. - Settings::WATER_DAMPENING);
        }
//        if (! stationary1)
        {
            p1_v = p1_v -
                (p1_r - p2_r) * (
                    dot(p1_v - p2_v_old, p1_r - p2_r)
                    / pow(distance(p1_r, p2_r), 2)
                    * 2 * m2 / (m1 + m2)
                );
            //*p1.v *= (1. - Settings::WATER_DAMPENING);
        }
//        if (! stationary1) {
//    //                    *p1.r += *p1.v * (Settings::dt - exactCollisionTime);//(1. - exactCollisionTime); //// ??
//    //                    *p1.r_former = *p1.r;
//        }
//        if (! stationary2) {
//    //                    *p2.r += *p2.v * (Settings::dt - exactCollisionTime);//(1. - exactCollisionTime); //// ??
//    //                    *p2.r_former = *p2.r;
//        }
    }

    rx1 = p1_r.x;
    ry1 = p1_r.y;
    rz1 = p1_r.z;
    vx1 = p1_v.x;
    vy1 = p1_v.y;
    vz1 = p1_v.z;
    rx2 = p2_r.x;
    ry2 = p2_r.y;
    rz2 = p2_r.z;
    vx2 = p2_v.x;
    vy2 = p2_v.y;
    vz2 = p2_v.z;
}

__global__ void collideAllKernel(unsigned particle_count,
                                 double *rx, double *ry, double *rz,
                                 double *vx, double *vy, double *vz,
                                 double *m,
                                 unsigned *stationary,
                                 double radius)
{
//    printf("Block %d, thread %d, block dim %d, grid dim %d.\n",
//           blockIdx.x, threadIdx.x, blockDim.x, gridDim.x);

//    for (unsigned i = 0; i < particle_count; ++i) {
//        for (unsigned j = i+1; j < particle_count; ++j) {
//            collidePair(rx[i], ry[i], rz[i], vx[i], vy[i], vz[i],
//                        rx[j], ry[j], rz[j], vx[j], vy[j], vz[j],
//                        m[i], m[j], radius, stationary[i], stationary[j]);
//        }
//    }

    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    //unsigned j = blockIdx.y * blockDim.y + threadIdx.y;
//    for (unsigned i = 0; i < particle_count; ++i)
        for (unsigned j = i+1; j < particle_count; ++j)
            collidePair(rx[i], ry[i], rz[i], vx[i], vy[i], vz[i],
                        rx[j], ry[j], rz[j], vx[j], vy[j], vz[j],
                        m[i], m[j], radius, stationary[i], stationary[j]);
}

void GPUCollideAll()
{
//    1152 cores
//    1024 threads per block
//    hipDeviceGet...(
//    CUDA Device Name           GeForce GTX 1060
//    Compute Capability         6.1
//    Processor Count            10
//    Cores per Processor        128
//    Threads per Multiprocessor 2048
//    Warp Size                  32 Threads

    int blocksPerGrid = Settings::PARTICLE_COUNT / 8;
    int threadsPerBlock = Settings::PARTICLE_COUNT / blocksPerGrid;
//    dim3 blocksPerGrid(512);          // 512 x 1 x 1
//    dim3 threadsPerBlock(1024, 1024); // 1024 x 1024 x 1

    std::vector<double> rx(Settings::PARTICLE_COUNT);
    std::vector<double> ry(Settings::PARTICLE_COUNT);
    std::vector<double> rz(Settings::PARTICLE_COUNT);
    std::vector<double> vx(Settings::PARTICLE_COUNT);
    std::vector<double> vy(Settings::PARTICLE_COUNT);
    std::vector<double> vz(Settings::PARTICLE_COUNT);
    std::vector<double> m(Settings::PARTICLE_COUNT);
    std::vector<unsigned> stationary(Settings::PARTICLE_COUNT);
    for (unsigned int i = 0; i < Settings::PARTICLE_COUNT; ++i) {
        rx[i]         = Particle::flows[0][i]->r->x;
        ry[i]         = Particle::flows[0][i]->r->y;
        rz[i]         = Particle::flows[0][i]->r->z;
        vx[i]         = Particle::flows[0][i]->v->x;
        vy[i]         = Particle::flows[0][i]->v->y;
        vz[i]         = Particle::flows[0][i]->v->z;
        m[i]          = Particle::flows[0][i]->m;
        stationary[i] = (unsigned) Particle::flows[0][i]->stationary;
    }

    double *rx_device = 0;
    hipMalloc((void**)&rx_device, rx.size() * sizeof(double));
    hipMemcpy(rx_device, rx.data(), rx.size() * sizeof(double), hipMemcpyHostToDevice);

    double *ry_device = 0;
    hipMalloc((void**)&ry_device, ry.size() * sizeof(double));
    hipMemcpy(ry_device, ry.data(), ry.size() * sizeof(double), hipMemcpyHostToDevice);

    double *rz_device = 0;
    hipMalloc((void**)&rz_device, rz.size() * sizeof(double));
    hipMemcpy(rz_device, rz.data(), rz.size() * sizeof(double), hipMemcpyHostToDevice);

    double *vx_device = 0;
    hipMalloc((void**)&vx_device, vx.size() * sizeof(double));
    hipMemcpy(vx_device, vx.data(), vx.size() * sizeof(double), hipMemcpyHostToDevice);

    double *vy_device = 0;
    hipMalloc((void**)&vy_device, vy.size() * sizeof(double));
    hipMemcpy(vy_device, vy.data(), vy.size() * sizeof(double), hipMemcpyHostToDevice);

    double *vz_device = 0;
    hipMalloc((void**)&vz_device, vz.size() * sizeof(double));
    hipMemcpy(vz_device, vz.data(), vz.size() * sizeof(double), hipMemcpyHostToDevice);

    double *m_device = 0;
    hipMalloc((void**)&m_device, m.size() * sizeof(double));
    hipMemcpy(m_device, m.data(), m.size() * sizeof(double), hipMemcpyHostToDevice);

    unsigned *stationary_device = 0;
    hipMalloc((void**)&stationary_device, stationary.size() * sizeof(unsigned));
    hipMemcpy(stationary_device, stationary.data(), stationary.size() * sizeof(unsigned), hipMemcpyHostToDevice);

    collideAllKernel<<<blocksPerGrid, threadsPerBlock>>>(
                                          Settings::PARTICLE_COUNT,
                                          rx_device, ry_device, rz_device,
                                          vx_device, vy_device, vz_device,
                                          m_device,
                                          stationary_device,
                                          Settings::PARTICLE_RADIUS);
    hipDeviceSynchronize();

    hipMemcpy(&rx[0], rx_device, rx.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&ry[0], ry_device, ry.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&rz[0], rz_device, rz.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&vx[0], vx_device, vx.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&vy[0], vy_device, vy.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&vz[0], vz_device, vz.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&m[0], m_device, m.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&stationary[0], stationary_device, stationary.size() * sizeof(unsigned), hipMemcpyDeviceToHost);

    for (unsigned i = 0; i < Settings::PARTICLE_COUNT; ++i) {
        Particle::flows[0][i]->r->x = rx[i];
        Particle::flows[0][i]->r->y = ry[i];
        Particle::flows[0][i]->r->z = rz[i];
        Particle::flows[0][i]->v->x = vx[i];
        Particle::flows[0][i]->v->y = vy[i];
        Particle::flows[0][i]->v->z = vz[i];
    }

    hipFree(rx_device);
    hipFree(ry_device);
    hipFree(rz_device);
    hipFree(vx_device);
    hipFree(vy_device);
    hipFree(vz_device);
    hipFree(m_device);
    hipFree(stationary_device);
}

#endif // CUDA_CU_H
