#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <builtin_types.h> // __global__ __host__ __device__

#define UNIFIED_MATH_CUDA_H
#include <vector_functions.h>
#include <math_functions.h>
#include <hip/hip_runtime.h> // hipMalloc(), hipMemcpy(), ...
#include <hip/hip_runtime_api.h>

//#include <thrust/device_vector.h>
//#include <thrust/host_vector.h>
//#include <thrust/system/cuda/experimental/pinned_allocator.h>

#include <vector>
//#ifdef COMPILER_MSVC
#include <algorithm>
//#endif

#include "machine/particle.h"
#include "physics/forces.h"
#include "physics/vector.h"
#include "util/settings.h"

#include <omp.h>

using namespace std;

#ifdef COMPILER_GPP
    #define LOOP_TYPE unsigned
#elif COMPILER_MSVC
    #define LOOP_TYPE int
#endif

#define CUDA_TIME_MEASUREMENT_INIT \
    hipEvent_t startEvent, stopEvent; \
    float time; \
    int bytes;

#define CUDA_TIME_MEASUREMENT_START \
    checkCudaErrors(hipEventCreate(&startEvent)); \
    checkCudaErrors(hipEventCreate(&stopEvent)); \
    checkCudaErrors(hipEventRecord(startEvent, 0));

#define CUDA_TIME_MEASUREMENT_END \
    checkCudaErrors(hipEventRecord(stopEvent, 0)); \
    checkCudaErrors(hipEventSynchronize(stopEvent)); \
    checkCudaErrors(hipEventElapsedTime(&time, startEvent, stopEvent));

#define CUDA_TIME_MEASUREMENT_FIN \
    checkCudaErrors(hipEventDestroy(startEvent)); \
    checkCudaErrors(hipEventDestroy(stopEvent));

////////////////////////////////////////////////////////////////////////////////
//// Interface for outside world. //////////////////////////////////////////////
//////// Kernel (= global = host + device) functions. //////////////////////////
////////////////////////////////////////////////////////////////////////////////

__forceinline __device__ double3 operator+(const double3 &v1, const double3 &v2) {
    return make_double3(v1.x+v2.x, v1.y+v2.y, v1.z+v2.z);
}

__forceinline __device__ double3 operator-(const double3 &v1, const double3 &v2) {
    return make_double3(v1.x-v2.x, v1.y-v2.y, v1.z-v2.z);
}

__forceinline __device__ double3 operator*(const double3 &v, const double &n) {
    return make_double3(v.x*n, v.y*n, v.z*n);
}

__forceinline __device__ double3 operator/(const double3 &v, const double &n) {
    return make_double3(v.x/n, v.y/n, v.z/n);
}

__forceinline __device__ double dot(const double3 &v1, const double3 &v2) {
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__forceinline __device__ double norm(const double3 &v) {
    //return sqrt(dot(v, v));
    return norm3d(v.x, v.y, v.z);
}

__forceinline __device__ double distance(const double3 &v1, const double3 &v2) {
    double3 v3 = v1 - v2;
    return norm(v3);
}

__forceinline __device__ double3 normal(const double3 &v) {
    return v / norm(v);
}

__forceinline __device__ void collidePair(double &rx1, double &ry1, double &rz1,
                                          double &vx1, double &vy1, double &vz1,
                                          double &rx2, double &ry2, double &rz2,
                                          double &vx2, double &vy2, double &vz2,
                                          double &m1, double &m2,
                                          double &radius,
                                          bool &is_stationary_p1,
                                          bool &is_stationary_p2)
{
    //double3 sphereNormal;
    double3 p1_r = make_double3(rx1, ry1, rz1);
    double3 p2_r = make_double3(rx2, ry2, rz2);
    double3 p1_v = make_double3(vx1, vy1, vz1);
    double3 p2_v = make_double3(vx2, vy2, vz2);
    double3 sphereNormal = normal(p1_r - p2_r);
    double distanceBorder = distance(p1_r, p2_r) - radius - radius;
    if (distanceBorder < 0.) {
        if (! is_stationary_p2)
        {
            p2_r = p2_r + sphereNormal * distanceBorder;
        }

        double3 p2_v_old = make_double3(p2_v.x, p2_v.y, p2_v.z);
        if (! is_stationary_p2)
        {
            p2_v = p2_v -
                (p2_r - p1_r) * (
                    dot(p2_v - p1_v, p2_r - p1_r)
                    / pow(distance(p2_r, p1_r), 2)
                    * 2 * m1 / (m1 + m2)
                );
    //                    *p2.v *= (1. - Settings::WATER_DAMPENING);
        }
        if (! is_stationary_p1)
        {
            p1_v = p1_v -
                (p1_r - p2_r) * (
                    dot(p1_v - p2_v_old, p1_r - p2_r)
                    / pow(distance(p1_r, p2_r), 2)
                    * 2 * m2 / (m1 + m2)
                );
            //*p1.v *= (1. - Settings::WATER_DAMPENING);
        }
        if (! is_stationary_p1) {
//    //                    *p1.r += *p1.v * (Settings::dt - exactCollisionTime);//(1. - exactCollisionTime); //// ??
//    //                    *p1.r_former = *p1.r;
        }
        if (! is_stationary_p2) {
//    //                    *p2.r += *p2.v * (Settings::dt - exactCollisionTime);//(1. - exactCollisionTime); //// ??
//    //                    *p2.r_former = *p2.r;
        }
    }

   // __syncthreads();
    rx1 = p1_r.x;
    ry1 = p1_r.y;
    rz1 = p1_r.z;
    vx1 = p1_v.x;
    vy1 = p1_v.y;
    vz1 = p1_v.z;
    rx2 = p2_r.x;
    ry2 = p2_r.y;
    rz2 = p2_r.z;
    vx2 = p2_v.x;
    vy2 = p2_v.y;
    vz2 = p2_v.z;
//    __syncthreads();
}

__global__ void collideAllKernel(unsigned particle_count,
                                 double *rx, double *ry, double *rz,
                                 double *vx, double *vy, double *vz,
                                 double *m,
                                 double radius,
                                 bool *stationary, int i)
{
//    for (unsigned i = 0; i < particle_count; ++i) {
//        for (unsigned j = i+1; j < particle_count; ++j) {
//            collidePair(rx[i], ry[i], rz[i], vx[i], vy[i], vz[i],
//                        rx[j], ry[j], rz[j], vx[j], vy[j], vz[j],
//                        m[i], m[j], radius, stationary[i], stationary[j]);
//        }
//    }

//    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
//    //unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
////    for (unsigned i = 0; i < particle_count; ++i)
//        for (unsigned j = i+1; j < particle_count; ++j)
//            collidePair(rx[i], ry[i], rz[i], vx[i], vy[i], vz[i],
//                        rx[j], ry[j], rz[j], vx[j], vy[j], vz[j],
//                        m[i], m[j], radius, stationary[i], stationary[j]);

//    printf("blockIdx.x %d, threadIdx.x %d, blockIdx.y %d, threadIdx.y %d, blockDim.x %d, blockDim.y %d, gridDim.x %d, gridDim.y %d.\n",
//           blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y, blockDim.x, blockDim.y, gridDim.x, gridDim.y);

//    unsigned i = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
//    unsigned j = i + blockIdx.x * blockDim.x + threadIdx.y;
//    for (unsigned int j = i+1; j < particle_count; ++j) {
    unsigned j = blockIdx.x * 100 + threadIdx.x * 10 + threadIdx.y;
    if (j > i)
        collidePair(rx[i], ry[i], rz[i], vx[i], vy[i], vz[i],
                    rx[j], ry[j], rz[j], vx[j], vy[j], vz[j],
                    m[i], m[j], radius, stationary[i], stationary[j]);
//    }

//    unsigned i = blockIdx.x;
//    unsigned j = blockIdx.y;
//    if (j > i)
//    {
//        collidePair(rx[i], ry[i], rz[i], vx[i], vy[i], vz[i],
//                    rx[j], ry[j], rz[j], vx[j], vy[j], vz[j],
//                    m[i], m[j], radius, stationary[i], stationary[j]);
//    }
}

void GPUCollideMallocs()
{
    static bool wasExecuted = false;
    if (! wasExecuted) {
        wasExecuted = true;

        // pinned host memory
        checkCudaErrors(hipHostMalloc((void**)&Particle::rx_host, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipHostMalloc((void**)&Particle::ry_host, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipHostMalloc((void**)&Particle::rz_host, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipHostMalloc((void**)&Particle::vx_host, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipHostMalloc((void**)&Particle::vy_host, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipHostMalloc((void**)&Particle::vz_host, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipHostMalloc((void**)&Particle::m_host, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipHostMalloc((void**)&Particle::is_stationary_host, Settings::PARTICLE_COUNT_2D * sizeof(bool)));

        checkCudaErrors(hipMalloc((void**)&Particle::rx_device, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipMalloc((void**)&Particle::ry_device, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipMalloc((void**)&Particle::rz_device, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipMalloc((void**)&Particle::vx_device, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipMalloc((void**)&Particle::vy_device, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipMalloc((void**)&Particle::vz_device, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipMalloc((void**)&Particle::m_device, Settings::PARTICLE_COUNT_2D * sizeof(double)));
        checkCudaErrors(hipMalloc((void**)&Particle::is_stationary_device, Settings::PARTICLE_COUNT_2D * sizeof(bool)));
    }
}

void GPUCollideFrees()
{
    // pinned host memory
    checkCudaErrors(hipHostFree(Particle::rx_host));
    checkCudaErrors(hipHostFree(Particle::ry_host));
    checkCudaErrors(hipHostFree(Particle::rz_host));
    checkCudaErrors(hipHostFree(Particle::vx_host));
    checkCudaErrors(hipHostFree(Particle::vy_host));
    checkCudaErrors(hipHostFree(Particle::vz_host));
    checkCudaErrors(hipHostFree(Particle::m_host));
    checkCudaErrors(hipHostFree(Particle::is_stationary_host));

    checkCudaErrors(hipFree(Particle::rx_device));
    checkCudaErrors(hipFree(Particle::ry_device));
    checkCudaErrors(hipFree(Particle::rz_device));
    checkCudaErrors(hipFree(Particle::vx_device));
    checkCudaErrors(hipFree(Particle::vy_device));
    checkCudaErrors(hipFree(Particle::vz_device));
    checkCudaErrors(hipFree(Particle::m_device));
    checkCudaErrors(hipFree(Particle::is_stationary_device));
}

void GPUCollideAll()
{    
    CUDA_TIME_MEASUREMENT_INIT

    CUDA_TIME_MEASUREMENT_START
    {
//        std::cout << Particle::rx_host[0] << std::endl << std::flush;
//        std::cout << Particle::flows[0][0]->r->x << std::endl << std::flush;
        #pragma omp parallel for if(Settings::PARALLEL_OMP)
        for (LOOP_TYPE i = 0; i < (LOOP_TYPE) Settings::PARTICLE_COUNT_2D; ++i) {
            Particle::rx_host[i]            = Particle::flows[0][i]->r->x;
            Particle::ry_host[i]            = Particle::flows[0][i]->r->y;
            Particle::rz_host[i]            = Particle::flows[0][i]->r->z;
            Particle::vx_host[i]            = Particle::flows[0][i]->v->x;
            Particle::vy_host[i]            = Particle::flows[0][i]->v->y;
            Particle::vz_host[i]            = Particle::flows[0][i]->v->z;
            Particle::m_host[i]             = Particle::flows[0][i]->m;
            Particle::is_stationary_host[i] = Particle::flows[0][i]->isStationary;
        }
    }
    CUDA_TIME_MEASUREMENT_END
    bytes = Settings::PARTICLE_COUNT_2D * (7 * sizeof(double) + 1 * sizeof(bool));
    printf("Host Vector to Host Vector bandwidth: %fGB / %fs = %fGB/s\n",
           bytes * 1e-9, time * 1e-3, (bytes * 1e-9) / (time * 1e-3));

    CUDA_TIME_MEASUREMENT_START
    {
        // pinned host memory
        checkCudaErrors(hipMemcpy(Particle::rx_device, Particle::rx_host, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(Particle::ry_device, Particle::ry_host, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(Particle::rz_device, Particle::rz_host, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(Particle::vx_device, Particle::vx_host, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(Particle::vy_device, Particle::vy_host, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(Particle::vz_device, Particle::vz_host, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(Particle::m_device, Particle::m_host, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(Particle::is_stationary_device, Particle::is_stationary_host, Settings::PARTICLE_COUNT_2D * sizeof(bool), hipMemcpyHostToDevice));
    }
    CUDA_TIME_MEASUREMENT_END
    bytes = Settings::PARTICLE_COUNT_2D * (7 * sizeof(double) + 1 * sizeof(bool));
    printf("Host to CUDA Device bandwidth: %fGB / %fs = %fGB/s\n",
           bytes * 1e-9, time * 1e-3, (bytes * 1e-9) / (time * 1e-3));

    CUDA_TIME_MEASUREMENT_START
    {
        //    hipDeviceGet...(
        //    1152 cores
        //    1024 threads per block
        //    Compute Capability         6.1
        //    Processor Count            10
        //    Cores per Processor        128
        //    Threads per Multiprocessor 2048
        //    Warp Size                  32 Threads
        //    Block has at least one warp, with at least 32 threads.

        for (unsigned i = 0; i < Settings::PARTICLE_COUNT_2D; ++i) {
//            int blocksPerGrid = Settings::PARTICLE_COUNT / 50;
//            int threadsPerBlock = Settings::PARTICLE_COUNT / blocksPerGrid;
            dim3 blocksPerGrid(20); // ? x ? x ?
            dim3 threadsPerBlock(10, 10); // ? x ? x ?
//            dim3 blocksPerGrid(2000, 2000); // ? x ? x ?
//            dim3 threadsPerBlock(1); // ? x ? x ?
            collideAllKernel<<<blocksPerGrid, threadsPerBlock>>>(
                Settings::PARTICLE_COUNT_2D,
                Particle::rx_device, Particle::ry_device, Particle::rz_device,
                Particle::vx_device, Particle::vy_device, Particle::vz_device,
                Particle::m_device,
                Settings::PARTICLE_RADIUS,
                Particle::is_stationary_device, i
            );
        }
        checkCudaErrors(hipDeviceSynchronize());
    }
    CUDA_TIME_MEASUREMENT_END
    printf("CUDA Collision Kernel execution time: %fs\n", time * 1e-3);

    CUDA_TIME_MEASUREMENT_START
    {
        // pinned host memory
        checkCudaErrors(hipMemcpy(&Particle::rx_host[0], Particle::rx_device, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&Particle::ry_host[0], Particle::ry_device, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&Particle::rz_host[0], Particle::rz_device, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&Particle::vx_host[0], Particle::vx_device, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&Particle::vy_host[0], Particle::vy_device, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&Particle::vz_host[0], Particle::vz_device, Settings::PARTICLE_COUNT_2D * sizeof(double), hipMemcpyDeviceToHost));
    }
    CUDA_TIME_MEASUREMENT_END
    bytes = Settings::PARTICLE_COUNT_2D * (6 * sizeof(double));
    printf("CUDA Device to Host bandwidth: %fGB / %fs = %fGB/s\n",
           bytes * 1e-9, time * 1e-3, (bytes * 1e-9) / (time * 1e-3));

    CUDA_TIME_MEASUREMENT_START
    {
        #pragma omp parallel for if(Settings::PARALLEL_OMP)
        for (LOOP_TYPE i = 0; i < (LOOP_TYPE) Settings::PARTICLE_COUNT_2D; ++i) {
            Particle::flows[0][i]->r->x = Particle::rx_host[i];
            Particle::flows[0][i]->r->y = Particle::ry_host[i];
            Particle::flows[0][i]->r->z = Particle::rz_host[i];
            Particle::flows[0][i]->v->x = Particle::vx_host[i];
            Particle::flows[0][i]->v->y = Particle::vy_host[i];
            Particle::flows[0][i]->v->z = Particle::vz_host[i];
        }
    }
    CUDA_TIME_MEASUREMENT_END
    bytes = Settings::PARTICLE_COUNT_2D * (6 * sizeof(double));
    printf("Host Vector to Host Vector bandwidth: %fGB / %fs = %fGB/s\n",
           bytes * 1e-9, time * 1e-3, (bytes * 1e-9) / (time * 1e-3));

    CUDA_TIME_MEASUREMENT_FIN
}
